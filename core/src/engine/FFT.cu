
#include <hip/hip_runtime.h>
#ifdef SPIRIT_USE_CUDA
#include "FFT.hpp"
#include <cufft.h>
namespace Engine
{
    namespace FFT
    {
        //Dont need the single transforms because cuFFT can do real batch transforms
        void Four_3D(FFT_cfg cfg, FFT_real_type * in, FFT_cpx_type * out)
        {
            std::cerr << "NOT IMPLEMENTED FOR cuFFT" << std::endl;
        }
        void iFour_3D(FFT_cfg cfg, FFT_cpx_type * in, FFT_real_type * out)
        {
            std::cerr << "NOT IMPLEMENTED FOR cuFFT" << std::endl;   
        }

        void batch_Four_3D(FFT_Plan & plan)
        {
            cufftExecR2C(plan.cfg, plan.real_ptr.data(), plan.cpx_ptr.data());
            cudaDeviceSynchronize();
        }

        void batch_iFour_3D(FFT_Plan & plan)
        {
            cufftExecC2R(plan.cfg, plan.cpx_ptr.data(), plan.real_ptr.data());
            cudaDeviceSynchronize();
        }

        void FFT_Plan::Create_Configuration()
        {
            int rank = this->dims.size();
            int *n = this->dims.data();
            int howmany = this->howmany;
            int istride = howmany, ostride = howmany;
            int *inembed = n, *onembed = n;
            
            int size = 1;
            for(auto k : dims)
            {
                size *= k;
            }
            int idist = 1, odist = 1;

            if(this->inverse == false)
            {
                cufftPlanMany(&this->cfg, rank, n, inembed, istride, idist, onembed, ostride, odist, CUFFT_R2C, howmany);
            } else 
            {
                cufftPlanMany(&this->cfg, rank, n, inembed, istride, idist, onembed, ostride, odist, CUFFT_C2R, howmany);
            }
        }

        void FFT_Plan::Free_Configuration()
        {
            cufftDestroy(this->cfg);
        }
    }
}
#endif